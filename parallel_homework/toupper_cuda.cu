#include <string.h>

#include <stdio.h>
#include <unistd.h>
#include <stdlib.h>
#include <ctype.h>
#include <sys/types.h>
#include <sys/mman.h>
#include <sys/stat.h>
#include <fcntl.h>
#include <omp.h>

#include "tbb/tick_count.h"
using tbb::tick_count;

char* map_file(char *filename, int *length_out) 
{
	struct stat file_stat;
	int fd = open(filename, O_RDWR);
	if (fd == -1) 
	{
		printf("failed to open file: %s\n", filename); 
		exit(1);
	}
	if (fstat(fd, &file_stat) != 0) 
	{
		printf("failed to stat file: %s\n", filename); 
		exit(1);
	}
	off_t length = file_stat.st_size;
	void *file = mmap(0, length, PROT_WRITE, MAP_SHARED, fd, 0);
	if (file == (void *)-1) 
	{
		printf("failed to stat file: %s\n", filename); 
		exit(1);
	}

	*length_out = length;
	return (char *)file;
}

int main(int argc, char *argv[]) 
{
	int length = 0;
	char *file = map_file(argv[1], &length);

	tick_count start = tick_count::now();

	// Your code here! (and maybe elsewhere)

	tick_count end = tick_count::now();

	printf("time = %f seconds\n", (end - start).seconds());  
}
