#include "hip/hip_runtime.h"
#include <string.h>

#include <stdio.h>
#include <unistd.h>
#include <stdlib.h>
#include <ctype.h>
#include <sys/types.h>
#include <sys/mman.h>
#include <sys/stat.h>
#include <fcntl.h>
#include <omp.h>

#include "tbb/tick_count.h"
using tbb::tick_count;

char* map_file(char *filename, int *length_out) 
{
	struct stat file_stat;
	int fd = open(filename, O_RDWR);
	if (fd == -1) 
	{
		printf("failed to open file: %s\n", filename); 
		exit(1);
	}
	if (fstat(fd, &file_stat) != 0) 
	{
		printf("failed to stat file: %s\n", filename); 
		exit(1);
	}
	off_t length = file_stat.st_size;
	void *file = mmap(0, length, PROT_WRITE, MAP_SHARED, fd, 0);
	if (file == (void *)-1) 
	{
		printf("failed to stat file: %s\n", filename); 
		exit(1);
	}

	*length_out = length;
	return (char *)file;
}


/*__global__ void cube(float * d_out, float * d_in){
	int idx=threadIdx.x;
    float f=d_in[idx];
    d_out[idx]=f*f*f;
}

int main(int argc, char ** argv) {
	const int ARRAY_SIZE = 64;
	const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);

	// generate the input array on the host
	float h_in[ARRAY_SIZE];
	for (int i = 0; i < ARRAY_SIZE; i++) {
		h_in[i] = float(i);
	}
	float h_out[ARRAY_SIZE];

	// declare GPU memory pointers
	float * d_in;
	float * d_out;

	// allocate GPU memory
	hipMalloc((void**) &d_in, ARRAY_BYTES);
	hipMalloc((void**) &d_out, ARRAY_BYTES);

	// transfer the array to the GPU
	hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);

	// launch the kernel
	cube<<<1, ARRAY_SIZE>>>(d_out, d_in);

	// copy back the result array to the CPU
	hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);

	// print out the resulting array
	for (int i =0; i < ARRAY_SIZE; i++) {
		printf("%f", h_out[i]);
		printf(((i % 4) != 3) ? "\t" : "\n");
	}

	hipFree(d_in);
	hipFree(d_out);

	return 0;
}*/
__global__ void upper(int length, char* file){

	int index=blockIdx.x * blockDim.x + threadIdx.x;
	while(index<length){
		if(file[index]>=97 && file[index]<=122){
			file[index]-=32;
		}
		index+=blockDim.x * gridDim.x;
	}

}
int main(int argc, char *argv[]) 
{
	int length = 0;
	char *file = map_file(argv[1], &length);
	//hipEvent_t start, stop;
	//hipEventCreate(&start);
	//hipEventCreate(&stop);
	
	//hipEventRecord(start);

	tick_count start = tick_count::now();

	// Your code here! (and maybe elsewhere)

	char* d_in;//gpu file pointer
	hipMalloc( (void**) &d_in, length+1);//not sure if need null terminator

	hipMemcpy(d_in, file, length+1, hipMemcpyHostToDevice);//copy file to device

	upper<<<2, 128>>>(length, d_in);
	
	hipMemcpy(file, d_in, length+1, hipMemcpyDeviceToHost);//copy modified file back to cpu
	//printf("%c\n", file[1]);


	tick_count end = tick_count::now();
	printf("time = %f seconds\n", (end - start).seconds());  
	
	hipFree(d_in);

	
	/*hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds=0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("time = %f milliseconds\n", milliseconds);
	*/

}
