#include "hip/hip_runtime.h"
#include <string.h>	

#include <stdio.h>
#include <unistd.h>
#include <stdlib.h>
#include <ctype.h>
#include <sys/types.h>
#include <sys/mman.h>
#include <sys/stat.h>
#include <fcntl.h>
#include <omp.h>

#include "tbb/tick_count.h" 
using tbb::tick_count;

char* map_file(char *filename, int *length_out) 
{
	struct stat file_stat;
	int fd = open(filename, O_RDONLY);
	if (fd == -1) 
	{
		printf("failed to open file: %s\n", filename); 
		exit(1);
	}
	if (fstat(fd, &file_stat) != 0) 
	{
		printf("failed to stat file: %s\n", filename); 
		exit(1);
	}
	off_t length = file_stat.st_size;
	void *file = mmap(0, length, PROT_WRITE, MAP_PRIVATE, fd, 0);
	if (file == (void *)-1) 
	{
		printf("failed to stat file: %s\n", filename); 
		exit(1);
	}

	*length_out = length;
	return (char *)file;
}
__global__ void count(int length, char* file, unsigned* d_1){
	int index=blockIdx.x * blockDim.x + threadIdx.x;
	while(index<length){
		d_1[ file[index] ]++;
		index+=blockDim.x * gridDim.x;
	}
}

int main(int argc, char *argv[]) 
{
	int length = 0;
	bool print = false;
	if (strncmp(argv[1], "-print", 7) == 0) 
	{
		print = true;
		argv = &argv[1];
	}
	char *file = map_file(argv[1], &length);
	unsigned histogram[256] = {0};

	tick_count start = tick_count::now();

	// Your code here! (and maybe elsewhere)
	unsigned* d_1;
	char* d_2;
	hipMalloc( (void**) &d_2, length+1);
	hipMalloc( (void**) &d_1, 256*sizeof(unsigned));
	
	hipMemcpy(d_1, histogram, 256*sizeof(unsigned), hipMemcpyHostToDevice);
	hipMemcpy(d_2, file, length+1, hipMemcpyHostToDevice);
	
	count<<<2, 128>>>(length, d_2, d_1);
	
	hipMemcpy(histogram, d_1, 256*sizeof(unsigned), hipMemcpyDeviceToHost);
	
	hipFree(d_1);
	hipFree(d_2);
	//not getting right values for some reason
	tick_count end = tick_count::now();

	printf("time = %f seconds\n", (end - start).seconds());  

	if (print) 
	{
		for (int i = 0 ; i < 128 ; i ++) 
		{
			if (histogram[i] != 0) 
			{
				printf("%c (%d): %d\n", i, i, histogram[i]);
			}
		}
	}
}
